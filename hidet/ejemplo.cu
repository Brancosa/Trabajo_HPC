#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

const int M = 1000;             // Tamaño población
const int initial_infected = 10; // Infectados iniciales
const float infection_prob = 1.0f; // Probabilidad infección (1.0 para que siempre infecte)

// Kernel CUDA para intentar infectar vecinos de cada infectado
__global__ void infectar_vecinos(
    bool* susceptibles_mask,
    bool* nuevos_infectados_mask,
    const int* infectados_indices,
    int num_infectados,
    int M,
    float infection_prob,
    unsigned long long seed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_infectados) return;

    int idx = infectados_indices[i];

    // Inicializar estado aleatorio per hilo
    hiprandState state;
    hiprand_init(seed, i, 0, &state);

    // Vecinos a infectar: idx-1 e idx+1
    int vecinos[2] = { idx - 1, idx + 1 };

    for (int j = 0; j < 2; ++j) {
        int vecino = vecinos[j];
        if (vecino >= 0 && vecino < M) {
            if (susceptibles_mask[vecino]) {
                float p = hiprand_uniform(&state);
                if (p < infection_prob) {
                    nuevos_infectados_mask[vecino] = true;
                }
            }
        }
    }
}

int main() {
    // Inicializar población en host
    std::vector<bool> susceptibles_mask(M, true);
    std::vector<int> infectados_indices;

    // Infectar inicialmente aleatoriamente
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, M - 1);
    while ((int)infectados_indices.size() < initial_infected) {
        int x = dis(gen);
        if (susceptibles_mask[x]) {
            susceptibles_mask[x] = false;
            infectados_indices.push_back(x);
        }
    }

    // Buffers device
    bool *d_susceptibles_mask, *d_nuevos_infectados_mask;
    int *d_infectados_indices;

    hipMalloc(&d_susceptibles_mask, M * sizeof(bool));
    hipMalloc(&d_nuevos_infectados_mask, M * sizeof(bool));

    int max_infectados = M; // máximo posible infectados

    hipMalloc(&d_infectados_indices, max_infectados * sizeof(int));

    int paso = 0;
    while (true) {
        // Copiar datos a device
        hipMemcpy(d_susceptibles_mask, susceptibles_mask.data(), M * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_infectados_indices, infectados_indices.data(), infectados_indices.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_nuevos_infectados_mask, 0, M * sizeof(bool));

        // Lanzar kernel
        int blockSize = 256;
        int numBlocks = (int)((infectados_indices.size() + blockSize - 1) / blockSize);
        infectar_vecinos<<<numBlocks, blockSize>>>(
            d_susceptibles_mask,
            d_nuevos_infectados_mask,
            d_infectados_indices,
            (int)infectados_indices.size(),
            M,
            infection_prob,
            1234ULL + paso);

        hipDeviceSynchronize();

        // Copiar nuevos infectados a host
        std::vector<bool> nuevos_infectados_mask(M);
        hipMemcpy(nuevos_infectados_mask.data(), d_nuevos_infectados_mask, M * sizeof(bool), hipMemcpyDeviceToHost);

        // Actualizar máscaras y lista de infectados en host
        int nuevos = 0;
        for (int i = 0; i < M; ++i) {
            if (nuevos_infectados_mask[i] && susceptibles_mask[i]) {
                susceptibles_mask[i] = false;
                infectados_indices.push_back(i);
                nuevos++;
            }
        }

        int susceptibles = 0;
        for (bool b : susceptibles_mask) if (b) susceptibles++;

        std::cout << "Paso " << paso << ": Infectados = " << infectados_indices.size() << " Susceptibles = " << susceptibles << std::endl;

        paso++;

        if (nuevos == 0 || susceptibles == 0) {
            std::cout << "No hay nuevos infectados o se infectó toda la población.\n";
            break;
        }
    }

    hipFree(d_susceptibles_mask);
    hipFree(d_nuevos_infectados_mask);
    hipFree(d_infectados_indices);

    return 0;
}
